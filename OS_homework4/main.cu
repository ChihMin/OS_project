#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include "file.h"

//1000 KB in global memory
#define STORAGE_SIZE 1085440
#define FCB_SIZE 36864
#define MAX_FILE_SIZE 1048576
#define MAX_FILE_NUMBER 1024
#define NAME_LENGTH 24 
#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

typedef unsigned char uchar ; 
typedef uint32_t u32 ; 
/*
typedef struct METADATA{
	u32 fp ; 
	u32 time ; 
	u32 size ;
	uchar fileName[24] ; 
} Meta ; 
*/

//secondary memory
__device__ __managed__ uchar *volume ; 
__device__ __managed__ Meta *metadata ; 

__global__ void mykernel( uchar *input, uchar *output ){
	//####kernel start####
	
	
	
	//####kernel end####
}

int main(){
	
	hipMallocManaged( &metadata, FCB_SIZE ) ; 
	hipMallocManaged( &volume, MAX_FILE_SIZE ) ; 
	init_volume() ; 

	uchar *input, *output ; 
	hipMallocManaged( &input, MAX_FILE_SIZE ) ; 
	hipMallocManaged( &output, MAX_FILE_SIZE ) ; 
	
	for(int i = 0; i < MAX_FILE_SIZE; ++i)
		output[i] = 0 ; 

	load_binaryFile( DATAFILE, input, MAX_FILE_SIZE ) ;

	hipSetDevice( 5 ) ; 	

	mykernel<<<1, 1>>>( input, output )  ;
	hipDeviceSynchronize() ; 
	write_binaryFile(OUTFILE, output, MAX_FILE_SIZE ) ;  
	hipDeviceReset() ;

	
	return 0 ; 
}
