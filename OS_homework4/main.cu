#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include "file.h"

// 1000 KB in global memory
#define STORAGE_SIZE 1085440

// 36 KB file control block
#define FCB_SIZE 36864

// file's maximum size 
#define MAX_FILE_SIZE 1048576

// maximum file number
#define MAX_FILE_NUMBER 1024

// file name at most 24(20) chars
#define NAME_LENGTH 24 

// G_READ mode is 0
#define G_READ 0 

// G_WRITE mode is 1
#define G_WRITE 1

// RM instruction = 0
#define RM 0

// list all file by size order 
#define LS_S 1

// list all file by modified time order
#define LS_D 2

#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

typedef unsigned char uchar ; 
typedef uint32_t u32 ; 
/*
typedef struct METADATA{
	u32 fp ; 
	u32 time ; 
	u32 size ;
	uchar fileName[24] ; 
} Meta ; 
*/

//secondary memory
__device__ __managed__ uchar *volume ; 
__device__ __managed__ Meta *metadata ; 
__device__ __managed__ u32 updated_at = 0 ; 
__device__ __managed__ u32 file_num = 0 ;
__device__ __managed__ u32 last_pos = 0 ;


const u32 FILE_OPEN_ERROR = (u32)-1 ; 


__global__ void mykernel( uchar *input, uchar *output ){
	//####kernel start####
	for(int i = 0; i < 10; ++i )
		volume[i] = 1 ; 
	
	u32 fp , status ; 

	fp = open("a.txt\0", G_WRITE ) ;
	status = write( input, 100, fp ) ;
	printf("status = %d\n",status ) ; 

	
	fp = open("b.txt\0", G_WRITE ) ;
	status = write( input, 100, fp ) ;
	printf("status = %d\n",status ) ; 

	for(int i = 0; i < file_num; ++i ){
		Meta *cur = &metadata[i] ; 
		printf("filename : %s\n", cur->fileName ) ;
		printf("size : %d\n", cur->size ); 
		printf("time : %d\n", cur->time ); 
		printf("fp : %d\n", cur->fp ); 
		printf("\n"); 
	}
	 
	
	fp = open("a.txt\0", G_WRITE ) ;
	status = write( input, 124, fp ) ;
	printf("status = %d\n",status ) ; 

	
	fp = open("c.txt\0", G_WRITE ) ;
	status = write( input, 124, fp ) ;
	printf("status = %d\n",status ) ; 
	
	fp = open("a.txt\0", G_WRITE ) ;
	status = write( input, 124, fp ) ;
	printf("status = %d\n",status ) ; 


	fp = open("b.txt\0", G_WRITE ) ;
	status = write( input, 999, fp ) ;
	printf("status = %d\n",status ) ; 
	
	gsys( LS_S ) ; 
	gsys( LS_D ) ;

	debug() ; 
		
	fp = open("a.txt\0", G_READ ) ; 
	if( fp == FILE_OPEN_ERROR )
		printf("file open error\n") ; 
	read( output, 100, fp ) ;  
	//####kernel end####
}

int main(){
	
	hipMallocManaged( &metadata, FCB_SIZE ) ; 
	hipMallocManaged( &volume, MAX_FILE_SIZE ) ; 
	init_volume() ; 

	uchar *input, *output ; 
	hipMallocManaged( &input, MAX_FILE_SIZE ) ; 
	hipMallocManaged( &output, MAX_FILE_SIZE ) ; 
	
	for(int i = 0; i < MAX_FILE_SIZE; ++i)
		output[i] = 0 ; 

	load_binaryFile( DATAFILE, input, MAX_FILE_SIZE ) ;

	hipSetDevice( 5 ) ; 	

	mykernel<<<1, 1>>>( input, output )  ;
	hipDeviceSynchronize() ; 
	write_binaryFile(OUTFILE, output, MAX_FILE_SIZE ) ;  
	hipDeviceReset() ;

	
	return 0 ; 
}
