#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include "file.h"

// 1000 KB in global memory
#define STORAGE_SIZE 1085440

// 36 KB file control block
#define FCB_SIZE 36864

// file's maximum size 
#define MAX_FILE_SIZE 1048576

// maximum file number
#define MAX_FILE_NUMBER 1024

// file name at most 24(20) chars
#define NAME_LENGTH 24 

// G_READ mode is 0
#define G_READ 0 

// G_WRITE mode is 1
#define G_WRITE 1


#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

typedef unsigned char uchar ; 
typedef uint32_t u32 ; 
/*
typedef struct METADATA{
	u32 fp ; 
	u32 time ; 
	u32 size ;
	uchar fileName[24] ; 
} Meta ; 
*/

//secondary memory
__device__ __managed__ uchar *volume ; 
__device__ __managed__ Meta *metadata ; 
__device__ __managed__ u32 updated_at = 0 ; 
__device__ __managed__ u32 file_num = 0 ;
__device__ __managed__ u32 last_pos = 0 ;


const u32 FILE_OPEN_ERROR = (u32)-1 ; 


__global__ void mykernel( uchar *input, uchar *output ){
	//####kernel start####
	for(int i = 0; i < 10; ++i )
		volume[i] = 1 ; 
	
	u32 fp = open("a.txt\0", G_WRITE ) ; 
	fp = open("a.txt\0", G_READ ) ; 
	if( fp == FILE_OPEN_ERROR )
		printf("file open error\n") ; 
	read( output + 1, 10, fp ) ;  
	for(int i = 0; i < 14; ++i )
		printf("%d\n",output[i] ) ;
	//####kernel end####
}

int main(){
	
	hipMallocManaged( &metadata, FCB_SIZE ) ; 
	hipMallocManaged( &volume, MAX_FILE_SIZE ) ; 
	init_volume() ; 

	uchar *input, *output ; 
	hipMallocManaged( &input, MAX_FILE_SIZE ) ; 
	hipMallocManaged( &output, MAX_FILE_SIZE ) ; 
	
	for(int i = 0; i < MAX_FILE_SIZE; ++i)
		output[i] = 0 ; 

	load_binaryFile( DATAFILE, input, MAX_FILE_SIZE ) ;

	hipSetDevice( 5 ) ; 	

	mykernel<<<1, 1>>>( input, output )  ;
	hipDeviceSynchronize() ; 
	write_binaryFile(OUTFILE, output, MAX_FILE_SIZE ) ;  
	hipDeviceReset() ;

	
	return 0 ; 
}
