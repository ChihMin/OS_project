#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include "file.h"

// 1000 KB in global memory
#define STORAGE_SIZE 1085440

// 36 KB file control block
#define FCB_SIZE 36864

// file's maximum size 
#define MAX_FILE_SIZE 1048576

// maximum file number
#define MAX_FILE_NUMBER 1024

// file name at most 24(20) chars
#define NAME_LENGTH 24 

// G_READ mode is 0
#define G_READ 0 

// G_WRITE mode is 1
#define G_WRITE 1

// RM instruction = 0
#define RM 0

// list all file by size order 
#define LS_S 1

// list all file by modified time order
#define LS_D 2

#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

typedef unsigned char uchar ; 
typedef uint32_t u32 ; 
/*
typedef struct METADATA{
	u32 fp ; 
	u32 time ; 
	u32 size ;
	uchar fileName[24] ; 
} Meta ; 
*/

//secondary memory
__device__ __managed__ uchar *volume ; 
__device__ __managed__ Meta *metadata ; 
__device__ __managed__ u32 updated_at = 0 ; 
__device__ __managed__ u32 file_num = 0 ;
__device__ __managed__ u32 last_pos = 0 ;


#define FILE_OPEN_ERROR -1 ; 


__global__ void mykernel( uchar *input, uchar *output ){
	//####kernel start####i

/*
	u32 fp ; 
	printf("max = %d\n",MAX_FILE_NUMBER ) ; 
	for(int i = 0; i < MAX_FILE_NUMBER; ++i){
		int tmp = i ;
		char s[30] ;
		int len = 0 ;
		do{
			s[len++] = tmp % 10 + '0' ; 
			tmp /= 10 ; 
		}while( tmp != 0 );
		

		s[len] = s[len+1] = '\0'  ; 
		fp = open( s, G_WRITE ) ; 
		fp = write( input, 1024, fp ) ; 
		//printf("%d -> %s\n", , s ) ; 
	}
	for(int i = 0; i < MAX_FILE_NUMBER; ++i){
		int tmp = i ;
		char s[30] ;
		int len = 0 ;
		do{
			s[len++] = tmp % 10 + '0' ; 
			tmp /= 10 ; 
		}while( tmp != 0 );
		

		s[len] = s[len+1] = '\0'  ; 
		gsys( RM , s ) ; 
		
		//printf("%d -> %s\n", , s ) ; 
	}
	

	printf("%d\n", fp) ;
	gsys( LS_S ) ;
*/	
	 
	//Sample TA Test Case 
	fp = open("t.txt\0", G_WRITE ) ;
	write(input, 64, fp ) ;
	
	fp = open("b.txt\0", G_WRITE ) ;
	write(input + 32, 32, fp ) ;

	fp = open("t.txt\0", G_WRITE); 
	write(input + 32, 32, fp );

	freeSpace() ; 
	
	fp = open("t.txt\0", G_READ) ;
	read(output, 32, fp ); 

	gsys(LS_D);
	gsys(LS_S);

	fp = open("b.txt\0", G_WRITE );
	write( input + 64, 12, fp );
	gsys(LS_S);
	gsys(LS_D);

	gsys(RM, "t.txt\0");
	gsys(LS_S) ;

	//####kernel end####
}

int main(){
	
	hipMallocManaged( &metadata, FCB_SIZE ) ; 
	hipMallocManaged( &volume, MAX_FILE_SIZE ) ; 
	init_volume() ; 

	uchar *input, *output ; 
	hipMallocManaged( &input, MAX_FILE_SIZE ) ; 
	hipMallocManaged( &output, MAX_FILE_SIZE ) ; 
	
	for(int i = 0; i < MAX_FILE_SIZE; ++i)
		output[i] = 0 ; 

	load_binaryFile( DATAFILE, input, MAX_FILE_SIZE ) ;

	hipSetDevice( 5 ) ; 	

	mykernel<<<1, 1>>>( input, output )  ;
	hipDeviceSynchronize() ; 
	write_binaryFile(OUTFILE, output, MAX_FILE_SIZE ) ;  
	hipDeviceReset() ;

	
	return 0 ; 
}
