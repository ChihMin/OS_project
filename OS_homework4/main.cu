#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include "file.h"

//1000 KB in global memory
#define STORAGE_SIZE 1085440

#define MAX_FILE_SIZE 1048576

#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

typedef unsigned char uchar ; 
typedef uint32_t u32 ; 

//secondary memory
__device__ __managed__ uchar *volume ; 

__global__ void mykernel( uchar *input, uchar *output ){
	//####kernel start####

	

	//####kernel end####
}

int main(){
	
	hipMallocManaged( &volume, STORAGE_SIZE ) ; 
	init_volume() ; 	

	for(int i = 0; i < STORAGE_SIZE; ++i)	
		printf("%d\n", volume[i] )  ; 
	
	uchar *input, *output ; 
	hipMallocManaged( &input, MAX_FILE_SIZE ) ; 
	hipMallocManaged( &output, MAX_FILE_SIZE ) ; 
	
	for(int i = 0; i < MAX_FILE_SIZE; ++i)
		output[i] = 0 ; 

	load_binaryFile( DATAFILE, input, MAX_FILE_SIZE ) ;

	hipSetDevice( 5 ) ; 	

	mykernel<<<1, 1>>>( input, output )  ;
	hipDeviceSynchronize() ; 
	write_binaryFile(OUTFILE, output, MAX_FILE_SIZE ) ; // i change here ~  
	hipDeviceReset() ;

	
	return 0 ; 
}
