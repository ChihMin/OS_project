#include "hip/hip_runtime.h"
#include "file.h"

extern __device__ __managed__ int PAGEFAULT ; 
extern __device__ __managed__ int inTime ; 
extern __device__ __managed__ uchar storage[] ; 

__device__ u32 paging( uchar *buffer, u32 frame_num, u32 offset ){

	u32 target ; 
	int pt_entries = PT_ENTRIES ;
	/* 
		The format of entry :
		1. Bit 0 is used to store valid/invalid bit 
		2. From bit 1 to 12 is used to store logical page number
		3. From bit 13 to 31 is used to store clock time
	*/
	/* 這裡是用來找有沒有重複 hit 的page*/ 
	for(int i = 0; i < pt_entries; ++i ){
		int mask = ( (1<<13) - 2 ) ; 
		int pageNum = ( pt[i] & mask ) >> 1 ; 
		/* pageNum 用來存pt[i]的logic page number */
		
		/* If frame_num(the logic page number want to query) 
			is the same as logical page number in entry   
		*/
		if( ( pt[i] & 1 )  && pageNum == frame_num ){
			int tmpTime = inTime++ ;
			// update hit time  	
			pt[i] = ( tmpTime << 13 ) | ( frame_num << 1 ) | 1 ; 
			return i * 32 + offset ;
		}
	}
	
	for(int i = 0; i < pt_entries; ++i ){
		if( pt[i] ^ 1 )	{	// If find invalid entry( empty entry )
			PAGEFAULT++ ;	// add PageFault
			/*
				update page table
			*/
			int tmpTime = inTime++ ; 
			pt[i] = ( tmpTime << 13 ) | ( frame_num << 1 ) | 1 ; 
			return i * 32 + offset  ; 
		}
	}

	int timeRange = -1 ; 
	// timeRange = CurrentTime - hitPageTime
	// timeRange is used to determine what the least time is 
	// if some time is earlier, the timeRange is wider
	// target variable is used to store the entry
	for(int i = 0; i < pt_entries; ++i ){
		int mask = -1 ; 
		int tmpTime  = (( mask << 13 ) &  pt[i] ) >> 13 ;
		int tmp = inTime - tmpTime ; 

		if( tmp > timeRange  ){
			target = i ;
			timeRange = tmp ;  
		}
	}
	
	PAGEFAULT++ ;
	/*
		move the page from shared memory to global memory 
		And move the page form secondary storage to shared memory 
	*/
	int mask = ( 1 << 13 ) - 2 ; 
	u32 tarFrame = ( pt[target] & mask) >> 1 ;	//要被換掉的logical page
	int beginAddress = tarFrame * 32; //要被換掉的page的目標secondary memory 
	for(int i = beginAddress, j = 0; j < 32; ++i , ++j){
		int sharedAddress = target * 32 + j ; // 當前要交換的physical memory address
		int curAddress = frame_num * 32 + j ; // 想要交換到physical memory address 的page
		 
		storage[i] = buffer[sharedAddress] ; 
		buffer[sharedAddress] = storage[curAddress]; 
	}
	pt[target] = ((inTime++) << 13 ) | ( frame_num << 1 ) | 1 ;
	return target * 32 + offset ;
}

__device__ void init_pageTable( int pt_entries ){
	for(int i = 0; i < pt_entries; ++i ){
		pt[i] = 0  ; 
	}
}

int load_binaryFile( const char *DATAFILE, uchar *input, int STORAGE_SIZE ){
	int size = 0 ; 
	uchar in ; 
	FILE *R = fopen( DATAFILE, "rb" ) ; 
	
	while( fread( &in, sizeof( uchar ), 1, R ) )	
		input[size++] = in ; 
	fclose( R )  ;
	
	return size ;  
}

void write_binaryFile( const char *OUTFILE, uchar *results, int input_size ){
	FILE *W = fopen( OUTFILE, "wb" ) ; 
	for(int i = 0; i < input_size; ++i ){
		fwrite( &results[i], sizeof( uchar ), 1, W ) ;
	}
	fclose( W ) ; 
}
